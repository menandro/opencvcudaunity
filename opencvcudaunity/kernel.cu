#include "hip/hip_runtime.h"
#include "opencvcudaunity.h"

// Initialize Cuda containers with specified width, height, and channels
// so that we don't have to allocate memory every call
int Ocu::initCuda(int width, int height, int channels) {
	this->width = width;
	this->height = height;
	this->channels = channels;

	// Compute stride based on width and stride alignment
	int m = this->StrideAlignment;
	int mod = this->width % m;
	if (mod)
		stride = this->width + m - mod;
	else
		stride = this->width;

	this->dataSize = stride * height * channels * sizeof(unsigned char);
	checkCudaErrors(hipMalloc(&d_input, dataSize));
	checkCudaErrors(hipMalloc(&d_output, dataSize));
	return 0;
}

int Ocu::blurCuda(unsigned char *input, unsigned char * output) {
	// Copy image from CPU(host) to GPU(device)
	checkCudaErrors(hipMemcpy(d_input, input, dataSize, hipMemcpyHostToDevice));

	// Blur
	Blur(d_input, d_output, width, height, stride);

	// Copy results from GPU to CPU
	checkCudaErrors(hipMemcpy(output, d_output, dataSize, hipMemcpyDeviceToHost));

	return 0;
}

// *******************************
// Kernel and Kernel Function Call
__global__
void BlurKernel(uchar4* input, uchar4 * output, int width, int height, int stride)
{
	const int ix = threadIdx.x + blockIdx.x * blockDim.x;
	const int iy = threadIdx.y + blockIdx.y * blockDim.y;

	const int pos = ix + iy * stride;

	int4 sum = make_int4(0, 0, 0, 0);
	int total = 0;
	if (ix >= width || iy >= height) return;
	for (int j = 0; j < 5; j++) {
		for (int i = 0; i < 5; i++) {
			int col = (ix + i - 2);
			int row = (iy + j - 2);
			if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
				sum.w += input[col + stride * row].w;
				sum.x += input[col + stride * row].x;
				sum.y += input[col + stride * row].y;
				sum.z += input[col + stride * row].z;
				total++;
			}
		}
	}
	output[pos].w = (uchar)(sum.w / total);
	output[pos].x = (uchar)(sum.x / total);
	output[pos].y = (uchar)(sum.y / total);
	output[pos].z = (uchar)(sum.z / total);
}

void Ocu::Blur(uchar4* input, uchar4 * output, int w, int h, int s) {
	dim3 threads(BlockWidth, BlockHeight);
	int blocksx = (w + threads.x - 1) / threads.x;
	int blocksy = (h + threads.y - 1) / threads.y;
	dim3 blocks(blocksx, blocksy);

	BlurKernel << <blocks, threads >> > (input, output, w, h, s);
}